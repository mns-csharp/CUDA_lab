#include "hip/hip_runtime.h"
////   test.cu   ////
#include "common.hpp"

const int min_ = 0;
const int max_ = 10;

/*
void OuterProduct(float* A, float* B, float** C, int N)
{
    for(int r=0 ; r<N ; r++)
    {
        for(int c=0 ; c<N ; c++)
        {
            for(int cc=0 ; cc<N ; cc++)
            {
                (*C)[r * N + c] += A[r * N + cc] * B[cc * N + c];
            }
        }
    }
}
*/

__global__ void MultiplyMatKernel(I* A, I* B, I* C, int N)
{
    int dimx = N;
	int dimy = N;
	int dimz = N;

    int r = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y * blockDim.y + threadIdx.y;
    int d = blockIdx.z * blockDim.z + threadIdx.z;

    if (r < N && c < N && d < N) 
	{
        int loc_c = d * dimx * dimy + c * dimx + r;
 
        for (int cc=0; cc<N; cc++) 
		{
		    int loc_a = (cc * dimx * dimy) + (c * dimx) + r;
		    int loc_b = (d * dimx * dimy) + (cc * dimx) + r;
            C[loc_c] += A[loc_a]*B[loc_b];
        }
    }
}


int main()
{
    I * device_a;
    I * device_b;
    I * device_c;
    I * host_a;
    I * host_b;
    I * host_c;
    int kernel_len;
	int length;
    dim3 threads_per_block;
    dim3 blocks_per_grid;    
    
	kernel_len = 2;
    length = kernel_len * kernel_len * kernel_len;
    host_a = (I *) malloc(sizeof(I) * length);
    host_b = (I *) malloc(sizeof(I) * length);
    host_c = (I *) malloc(sizeof(I) * length);
    
    if (host_a == nullptr || host_b == nullptr || host_c == nullptr)
    {
        std::cerr << "Error: Memory allocation for host arrays failed." << std::endl;
        exit(1);
    }

    CHECK_CUDA_ERROR(hipMalloc((void**) &device_a, sizeof(I) * length));
    CHECK_CUDA_ERROR(hipMalloc((void**) &device_b, sizeof(I) * length));
    CHECK_CUDA_ERROR(hipMalloc((void**) &device_c, sizeof(I) * length));

    for (int i = 0; i < length ; ++i) 
    {
        host_a[i] = rand_float(min_, max_);
        host_b[i] = rand_float(min_, max_);
        host_c[i] = 0;
    }

    //int max_thread = 1024;
    int max_block = 2048;
    threads_per_block = dim3(32, 8, 4); // because, 1204 = 32*8*4 
    blocks_per_grid = dim3(max_block, max_block, max_block); 

    print_dim3("threads_per_block", threads_per_block);
    print_dim3("blocks_per_grid", blocks_per_grid);

    CHECK_CUDA_ERROR(hipMemcpy(device_a, host_a, sizeof(I) * length, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(device_b, host_b, sizeof(I) * length, hipMemcpyHostToDevice));
    
    MultiplyMatKernel<<<blocks_per_grid, threads_per_block>>>(device_a, device_b, device_c, kernel_len);
	
    CHECK_LAST_CUDA_ERROR();	
    CHECK_CUDA_ERROR(hipDeviceSynchronize());	
    CHECK_CUDA_ERROR(hipMemcpy(host_c, device_c, sizeof(I) * length, hipMemcpyDeviceToHost));
	
    write_output_to_file(host_a, host_b, host_c, "output.txt", length);
	
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
	
    free(host_a);
    free(host_b);
    free(host_c);
}

