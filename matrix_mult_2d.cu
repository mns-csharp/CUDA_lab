#include "hip/hip_runtime.h"
////   test.cu   ////
#include "common.hpp"

const int min_ = 0;
const int max_ = 10;

/*
void OuterProduct(float* A, float* B, float** C, int N)
{
    for(int r=0 ; r<N ; r++)
    {
        for(int c=0 ; c<N ; c++)
        {
            for(int cc=0 ; cc<N ; cc++)
            {
                (*C)[r * N + c] += A[r * N + cc] * B[cc * N + c];
            }
        }
    }
}
*/

__global__ void MultiplyMatKernel(I* A, I* B, I* C, int N)
{
    int dimx = N;
	int dimy = N;
	int dimz = N;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < N && j < N && k < N) 
	{
        int loc_c = k * dimx * dimy + j * dimx + i;
        int loc_a = k * dimx * dimy + j * dimx + i;
        int loc_b = k * dimx * dimy + j * dimx + i;
 
        for (int l=0; l<N; l++) 
		{
            C[loc_c] += A[loc_a+l]*B[loc_b+l];
        }
    }
}

int main()
{
    I * device_a;
    I * device_b;
    I * device_c;
    I * host_a;
    I * host_b;
    I * host_c;
    int kernel_len;
	int length;
    dim3 threads_per_block;
    dim3 blocks_per_grid;    
    
	kernel_len = 2;
    length = kernel_len * kernel_len * kernel_len;
    host_a = (I *) malloc(sizeof(I) * length);
    host_b = (I *) malloc(sizeof(I) * length);
    host_c = (I *) malloc(sizeof(I) * length);
    
    if (host_a == nullptr || host_b == nullptr || host_c == nullptr)
    {
        std::cerr << "Error: Memory allocation for host arrays failed." << std::endl;
        exit(1);
    }

    CHECK_CUDA_ERROR(hipMalloc((void**) &device_a, sizeof(I) * length));
    CHECK_CUDA_ERROR(hipMalloc((void**) &device_b, sizeof(I) * length));
    CHECK_CUDA_ERROR(hipMalloc((void**) &device_c, sizeof(I) * length));

    for (int i = 0; i < length ; ++i) 
    {
        host_a[i] = rand_float(min_, max_);
        host_b[i] = rand_float(min_, max_);
        host_c[i] = 0;
    }

    //int max_thread = 1024;
    int max_block = 62500;
    threads_per_block = dim3(32, 8, 4); // because, 1204 = 32*8*4 
    blocks_per_grid = dim3(max_block, max_block, max_block); 

    print_dim3("threads_per_block", threads_per_block);
    print_dim3("blocks_per_grid", blocks_per_grid);

    CHECK_CUDA_ERROR(hipMemcpy(device_a, host_a, sizeof(I) * length, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(device_b, host_b, sizeof(I) * length, hipMemcpyHostToDevice));
    
    MultiplyMatKernel<<<blocks_per_grid, threads_per_block>>>(device_a, device_b, device_c, 100);
	
    CHECK_LAST_CUDA_ERROR();	
    CHECK_CUDA_ERROR(hipDeviceSynchronize());	
    CHECK_CUDA_ERROR(hipMemcpy(host_c, device_c, sizeof(I) * length, hipMemcpyDeviceToHost));
	
    write_output_to_file(host_a, host_b, host_c, "output.txt", length);
	
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
	
    free(host_a);
    free(host_b);
    free(host_c);
}

