#include "hip/hip_runtime.h"
////   test.cu   ////
#include "common.hpp"

const int min_ = 0;
const int max_ = 10;

__global__ void MultiplyMatKernel(I* A, I* B, I* C, int N)
{
    int dimx = N;
	int dimy = N;
	int dimz = N;

    int r = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y * blockDim.y + threadIdx.y;
	int d = blockIdx.z * blockDim.z + threadIdx.z;

    if (r < N && c < N && d < N) 
	{
        int loc_c = d * dimx * dimy + c * dimx + r;
		int loc_a = d * dimx * dimy + c * dimx + r;
		int loc_b = d * dimx * dimy + c * dimx + r;
        for (int cc=0; cc<N; cc++) 
		{	
            C[loc_c] += A[loc_a+cc]*B[loc_b+cc];
        }
		printf("C[%d]=%d  \n", loc_c, C[loc_c]);
    }
}

void Transpose(float *A, float**At, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            // copy the value at (i,j) to (j,i) in At
            (*At)[j*N + i] = A[i*N + j];
        }
    }
}

int main()
{    
    I * host_a;
    I * host_b;
	I * host_b_T;
    I * host_c;
    I * device_a;
    I * device_b;
    I * device_c;
	int kernel_len;
	int length;
    dim3 threads_per_block;
    dim3 blocks_per_grid;    
    
	kernel_len = 3;
    length = kernel_len * kernel_len * 1;
    host_a = (I *) malloc(sizeof(I) * length);
    host_b = (I *) malloc(sizeof(I) * length);
	host_b_T = (I *) malloc(sizeof(I) * length);
    host_c = (I *) malloc(sizeof(I) * length);
    
    if (host_a == nullptr || host_b == nullptr || host_c == nullptr)
    {
        std::cerr << "Error: Memory allocation for host arrays failed." << std::endl;
        exit(1);
    }

    CHECK_CUDA_ERROR(hipMalloc((void**) &device_a, sizeof(I) * length));
    CHECK_CUDA_ERROR(hipMalloc((void**) &device_b, sizeof(I) * length));
    CHECK_CUDA_ERROR(hipMalloc((void**) &device_c, sizeof(I) * length));

    for (int i = 0; i < length ; ++i) 
    {
        host_a[i] = i+1;
        host_b[i] = i+1;
        host_c[i] = 0;
    }

	Transpose(host_b, host_b_T, kernel_len);

    int dimx = kernel_len;
    int dimy = kernel_len;
    int dimz = 1;

    //int max_thread = 1024;
    threads_per_block = dim3(32, 8, 4); // because, 1204 = 32*8*4 
    blocks_per_grid = dim3((dimx + threads_per_block.x - 1) / threads_per_block.x, 
	                       (dimy + threads_per_block.y - 1) / threads_per_block.y, 
						   (dimz + threads_per_block.z - 1) / threads_per_block.z);

    print_dim3("threads_per_block", threads_per_block);
    print_dim3("blocks_per_grid", blocks_per_grid);

    CHECK_CUDA_ERROR(hipMemcpy(device_a, host_a, sizeof(I) * length, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(device_b, host_b_T, sizeof(I) * length, hipMemcpyHostToDevice));
    
    MultiplyMatKernel<<<blocks_per_grid, threads_per_block>>>(device_a, device_b, device_c, kernel_len);
	
    CHECK_LAST_CUDA_ERROR();	
    CHECK_CUDA_ERROR(hipDeviceSynchronize());	
    CHECK_CUDA_ERROR(hipMemcpy(host_c, device_c, sizeof(I) * length, hipMemcpyDeviceToHost));
	
    write_output_to_file(host_a, host_b, host_c, "output.txt", length);
	
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
	
    free(host_a);
    free(host_b);
    free(host_c);
}

