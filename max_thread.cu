#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        printf("Device %d: %s\n", i, deviceProp.name);
        printf("  Maximum threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    }
    return 0;
}
