#include "hip/hip_runtime.h"
#include "include/template.hpp"

const int min = 0;
const int min = 10;

__host__ void init_data_(CudaManager<float>& manager) 
{
    for (int i = 0; i < manager.get_length(); ++i) 
	{
        manager.set_host_a(i, rand_int(min, max));
        manager.set_host_b(i, rand_int(min, max));
        manager.set_host_c(i, rand_int(min, max));
    }
}

__global__ void vector_add(float *arr1, float *arr2, float *outp, int n) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) 
    {
        outp[tid] = arr1[tid] + arr2[tid];
    }
}

int main()
{
    init_rand();
	
    CudaManager<float> manager;
    manager.allocate_mem(10);
    manager.init_data(init_data_);
    manager.display_host();
	manager.launch_kernel(vector_add);
    manager.display_host();
    manager.free_mem();
}


