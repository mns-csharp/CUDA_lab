#include "hip/hip_runtime.h"
#include "template.hpp"

const int min_ = 0;
const int max_ = 10;

__host__ void init_data_(CudaManager<float>& manager) 
{
    for (int i = 0; i < manager.get_length(); ++i) 
	{
        manager.set_host_a(i, rand_int(min_, max_));
        manager.set_host_b(i, rand_int(min_, max_));
        manager.set_host_c(i, 0);
    }
}

__global__ void vector_add(float *arr1, float *arr2, float *outp, int length) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < length) 
    {
        outp[tid] = arr1[tid] + arr2[tid];
    }
}

int main()
{
    init_rand();
	
    CudaManager<float> manager;
    manager.allocate_mem(1000000);
    manager.init_data(init_data_);
	manager.set_thread_dim(16, 16, 1);
    //manager.display_host_data();
	manager.launch_kernel(vector_add);
	manager.display_elapsed_time();
    manager.write_output_to_file("template_output.txt");
    manager.free_mem();
}


