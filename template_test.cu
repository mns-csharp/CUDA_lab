#include "hip/hip_runtime.h"
#include "template.hpp"

__host__ void init_data_func(CudaManager<float>& manager) 
{
    for (int i = 0; i < manager.length; ++i) {
        manager.host_a[i] = rand();
        manager.host_b[i] = rand();
        manager.host_c[i] = rand();
    }
}

__global__ void vector_add(float *arr1, float *arr2, float *outp, int n) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) 
    {
        outp[tid] = arr1[tid] + arr2[tid];
    }
}

int main()
{
    CudaManager<float> manager;
    manager.allocate_mem(10);
    manager.init_data(init_data_func);
    manager.launch_kernel(vector_add);
    manager.display_data();
    manager.free_mem();
}
